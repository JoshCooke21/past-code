
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <math.h>
#include <chrono>

const double pi = 4.0 * atan(1.0);
__constant__ double c_pi;
__constant__ double c_spacing;
__constant__ int c_num_dipoles;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

// define wrapper functions giving file and line number of error
#define cuda_safe_call( err ) __cuda_safe_call( err, __FILE__, __LINE__ )
#define cuda_check_error() __cuda_check_error( __FILE__, __LINE__ )

inline void __cuda_safe_call(hipError_t err,const char *file,const int line){
  // only produce code if error checking enabled
  #ifdef CUDA_ERROR_CHECK
  if ( hipSuccess != err ){ // check for error
    // print out error message
    std::cerr << "cuda_safe_call() failed at " << file << ":" << line
    << " " << hipGetErrorString( err ) << std::endl;
    // exit code with error code -1
    exit( -1 );
  }
  #endif
  return;
}

class vtimer_t{

private:
   std::chrono::high_resolution_clock::time_point start_time;
   std::chrono::high_resolution_clock::time_point end_time;

public:
   // start the timer
   void start(){
      start_time = std::chrono::high_resolution_clock::now();
   }

   // start the timer
   void stop(){
      end_time = std::chrono::high_resolution_clock::now();
   }

   // get the elapsed time in milliseconds
   double elapsed_time(){

      // work out elapsed time
      return 1.e-9*double(std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count());

   }
};



__global__
void kernel_function(double *dev_x_coords, double *dev_y_coords, double *dev_z_coords,
                     double *dev_unit_mu, double *dev_b_x, double *dev_b_y, double *dev_b_z){

  //This array will hold the B field contributions from every dipole onto A per thread
  // to then be summed on threadIdx=0 and entered into the dev_b_(x,y,z) vector,
  // which holds the B field value at every dipole
  extern __shared__ double shared_b[];

  //The outer for loop will be governed by the number of blocks
  //The inner for loop                     the number of threads
  int outer_start  = blockIdx.x;
  int outer_stride = gridDim.x;

  int inner_start  = threadIdx.x;
  int inner_stride = blockDim.x;

  //Pull constants into local memory
  int d_num_dipoles = c_num_dipoles;
  double d_spacing = c_spacing;
  double d_pi = c_pi;

  if (threadIdx.x==0 && blockIdx.x==0) printf("Running with %d blocks and %d threads/block \n", gridDim.x, blockDim.x);

  for (int i=outer_start; i<d_num_dipoles; i+=outer_stride){ // Loop over dipole A

    double coords_i[3]; //Stores the position of dipole A
    coords_i[0] = dev_x_coords[i];
    coords_i[1] = dev_y_coords[i];
    coords_i[2] = dev_z_coords[i];

    //Each thread has its own version of addition that it keeps adding to
    //  so that the sum of addition across all threads in a block gives
    //  the magnetic field at that dipole
    double addition[3]={};

    //Calculate the self-interaction on tid=0 so it only gets added once
    if(threadIdx.x==0){
      double spacing3 = d_spacing*d_spacing*d_spacing;
      for (int n=0; n<3; n++) addition[n] += 2.0 * dev_unit_mu[n] / (3.0 * spacing3 );
    }

    for (int j=inner_start; j<d_num_dipoles; j+=inner_stride){ // Loop over dipole B

      if (i!=j){ // Non-self Interaction

        double disp_vec[3] = {dev_x_coords[j] - coords_i[0], dev_y_coords[j] - coords_i[1], dev_z_coords[j] - coords_i[2]};

        double vec_length = sqrt(disp_vec[0]*disp_vec[0] + disp_vec[1]*disp_vec[1] + disp_vec[2]*disp_vec[2]);
        double vec_length3 = vec_length*vec_length*vec_length;

        double unit_disp_vec[3]={};
        for (int n=0; n<3; n++){ unit_disp_vec[n] = disp_vec[n] / vec_length; }

        double dot_product = unit_disp_vec[0]*dev_unit_mu[0] + unit_disp_vec[1]*dev_unit_mu[1] + unit_disp_vec[2]*dev_unit_mu[2];

        for (int n=0; n<3; n++){
          addition[n] += 0.25 * (1.0/d_pi) * (3.0 * unit_disp_vec[n] * dot_product - dev_unit_mu[n] ) / (vec_length3);
        }

      } // Self-interaction is already accounted for on threadId 0

    } // Close dipole B loop

    //Ensure all threads have finished loop 2
    __syncthreads();

    //Put the contributions from each thread into the shared array
    shared_b[threadIdx.x] = addition[0];
    shared_b[threadIdx.x+blockDim.x] = addition[1];
    shared_b[threadIdx.x+2*blockDim.x] = addition[2];

    if (threadIdx.x==0){

      //Sum up the contributions from all thread calculations
      double dipole_mag_x = 0.0;
      double dipole_mag_y = 0.0;
      double dipole_mag_z = 0.0;
      for (int n=0; n<blockDim.x; n++) dipole_mag_x += shared_b[n];
      for (int n=0; n<blockDim.x; n++) dipole_mag_y += shared_b[blockDim.x+n];
      for (int n=0; n<blockDim.x; n++) dipole_mag_z += shared_b[2*blockDim.x+n];

      //Store the magnetic field at dipole A
      dev_b_x[i] = dipole_mag_x;
      dev_b_y[i] = dipole_mag_y;
      dev_b_z[i] = dipole_mag_z;

    } // Close if threadId==0

  } // Close dipole A loop

  //Ensure all threads have finished all calculations
  __syncthreads();
}


int main(int argc, char* argv[]){

  vtimer_t timer;
  timer.start();

  std::vector<double> unit_mag_moment = {0.0 , 0.0 , 1.0}; // Set magnetic moment direction, currently in z_hat
  std::vector<double> total_mag(3, 0.0); // Initialize the total magnetic field of the ovoid

  // Ellipsoid parameters
  const double r_x=100.0, r_y=100.0, r_z=200.0; // Ellipsoid radii, 10nm and 20nm
  const double spacing=3; // Spacing of points (3 Angstroms)

  //These arrays store the coordinates of the dipoles inside the ellipsoid
  std::vector<double> x_coords(0), y_coords(0), z_coords(0);

  // Looping over z first means that the centre-plane dipoles are grouped together in the coordinate vectors
  for (int k=-ceil(r_z/spacing); k<(ceil(r_z/spacing)+1); k++){ // Looping over dipoles in z
    double z = double(k)*spacing;
    for (int i=-ceil(r_x/spacing); i<(ceil(r_x/spacing)+1); i++){ // Looping over dipoles in x
      double x = double(i)*spacing;
      for (int j=-ceil(r_y/spacing); j<(ceil(r_y/spacing)+1); j++){ // Looping over dipoles in y
        double y = double(j)*spacing;

        if ( x*x/(r_x*r_x) + y*y/(r_y*r_y) + z*z/(r_z*r_z) <= 1){
          x_coords.push_back(x);
          y_coords.push_back(y);
          z_coords.push_back(z);
        }

      } // Close k
    } // Close j
  } // Close i

  //All three coordinate arrays have the same size
  int num_dipoles = x_coords.size();
  std::cout << "Number of dipoles inside: " << num_dipoles << std::endl;
  std::cout << std::endl;

  //These vectors will store the values of the magnetic field at each dipole
  std::vector<double> b_x_alldipoles(num_dipoles, 0.0);
  std::vector<double> b_y_alldipoles(num_dipoles, 0.0);
  std::vector<double> b_z_alldipoles(num_dipoles, 0.0);

  //Allocate arrays on device
  printf("Allocating arrays on device\n");
  int bytes = num_dipoles*sizeof(double); //Same for all three x,y,z arrays
  double *dev_x_coords, *dev_y_coords, *dev_z_coords, *dev_unit_mu, *dev_b_x, *dev_b_y, *dev_b_z;
  cuda_safe_call(hipMalloc(&dev_x_coords, bytes));
  cuda_safe_call(hipMalloc(&dev_y_coords, bytes));
  cuda_safe_call(hipMalloc(&dev_z_coords, bytes));
  cuda_safe_call(hipMalloc(&dev_unit_mu, 3*sizeof(double)));
  cuda_safe_call(hipMalloc(&dev_b_x, bytes));
  cuda_safe_call(hipMalloc(&dev_b_y, bytes));
  cuda_safe_call(hipMalloc(&dev_b_z, bytes));

  //Copy data from host to device
  printf("Copying data to device\n");
  cuda_safe_call(hipMemcpy(dev_x_coords, x_coords.data(), bytes, hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_y_coords, y_coords.data(), bytes, hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_z_coords, z_coords.data(), bytes, hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_unit_mu, unit_mag_moment.data(), 3*sizeof(double), hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_b_x, b_x_alldipoles.data(), bytes, hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_b_y, b_y_alldipoles.data(), bytes, hipMemcpyHostToDevice));
  cuda_safe_call(hipMemcpy(dev_b_z, b_z_alldipoles.data(), bytes, hipMemcpyHostToDevice));

  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(c_spacing), &spacing, sizeof(double)));
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(c_num_dipoles), &num_dipoles, sizeof(int)));
  cuda_safe_call(hipMemcpyToSymbol(HIP_SYMBOL(c_pi), &pi, sizeof(double)));

  //Run the parallelized section
  int num_threads_in_block = std::stoi(argv[1]);
  int num_blocks = std::stoi(argv[2]);

  //This sets the number of bytes for the shared array
  int total_shared_bytes = 3*num_threads_in_block*sizeof(double);

  std::cout << "Beginning calculation." << std::endl;

  kernel_function<<<num_blocks, num_threads_in_block, total_shared_bytes>>>(dev_x_coords, dev_y_coords, dev_z_coords, dev_unit_mu,
                                                                            dev_b_x, dev_b_y, dev_b_z);

  //Ensure the CPU and GPU are at the same point (GPU finished, CPU waiting)
  hipDeviceSynchronize();
  std::cout << "Calculation complete." << std::endl << std::endl;

  //Copy data from device to host
  cuda_safe_call(hipMemcpy(&b_x_alldipoles[0], dev_b_x, bytes, hipMemcpyDeviceToHost));
  cuda_safe_call(hipMemcpy(&b_y_alldipoles[0], dev_b_y, bytes, hipMemcpyDeviceToHost));
  cuda_safe_call(hipMemcpy(&b_z_alldipoles[0], dev_b_z, bytes, hipMemcpyDeviceToHost));

  printf("Copied data back from device \n");

  std::ofstream dfile;
  dfile.open("centre_plane_p.dat");
  dfile << "# x, y, B_z" << std::endl;

  //These are needed to return B to SI units
  const double mu   = 9.274*1E-24;
  const double mu_0 = 4.0*pi*1E-7;

  //Since all the z=0 points are grouped, I use new_x and old_x to check when the x-coordinate changes
  //This allows the 2D plot to plot correctly
  printf("Writing to centre_plane.dat\n");
  double old_x = 0;
  for (int i=0; i<num_dipoles; i++){
    if (z_coords[i]==0){
      double new_x = x_coords[i];

      if (old_x!=new_x) dfile << std::endl; // The newline allows the plotting
      // If on centre plane of ellipsoid, write x and y coords and z-component of dipole_mag
      dfile << x_coords[i] << " " << y_coords[i] << " " << b_z_alldipoles[i]* mu*mu_0*1E30 << std::endl;

      old_x = new_x;
    }
  }

  dfile.close();
  printf("Finished writing to centre_plane.dat\n");

  printf("Totalling the magnetic field values\n");
  for (int n=0; n<num_dipoles; n++){
    total_mag[0]+=b_x_alldipoles[n];
    total_mag[1]+=b_y_alldipoles[n];
    total_mag[2]+=b_z_alldipoles[n];
  }

    // Calculate average magnetisation in all 3 directions (store as vector)
  printf("Calculating the demagnetizing factor\n\n");
  std::vector<double> demagnet(3,0.0);
  std::cout << "Average B, Demagnet. factor" << std::endl;
  
  for (int n=0; n<3; n++){
    double average_mag = total_mag[n] / x_coords.size();
    demagnet[n] = 1.0 - pow(spacing,3) * average_mag;
    std::cout << average_mag* mu*mu_0*1E30 << " " << demagnet[n] << std::endl;;
  }
  std::cout << std::endl; // Prints all vector elements on the same line


  hipFree(dev_x_coords);
  hipFree(dev_y_coords);
  hipFree(dev_z_coords);
  hipFree(dev_unit_mu);
  hipFree(dev_b_x);
  hipFree(dev_b_y);
  hipFree(dev_b_z);

  timer.stop();
  // printf("%d %d %f \n", num_blocks, num_threads_in_block, timer.elapsed_time());
  std::cout << "Calculation complete. Total elapsed time: " << timer.elapsed_time() << std::endl;

  return 0;

}
